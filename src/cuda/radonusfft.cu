#include "hip/hip_runtime.h"
#include "radonusfft.cuh"
#include "kernels_radonusfft.cuh"
#include <stdio.h>

radonusfft::radonusfft(size_t N_, size_t Ntheta_, size_t Nz_, float center_)
{
	N = N_;
	Ntheta = Ntheta_;
	Nz = Nz_;
	center = center_;
	float eps = 1e-6;
	mu = -log(eps)/(2*N*N);
	M = ceil(2*N*1/PI*sqrt(-mu*log(eps)+(mu*N)*(mu*N)/4));
	hipMalloc((void**)&f,N*N*Nz*sizeof(float2));
	hipMalloc((void**)&g,N*Ntheta*Nz*sizeof(float2));
	hipMalloc((void**)&fde,(2*N+2*M)*(2*N+2*M)*Nz*sizeof(float2));
	hipMalloc((void**)&x,N*Ntheta*sizeof(float));
	hipMalloc((void**)&y,N*Ntheta*sizeof(float));
	hipMalloc((void**)&theta,Ntheta*sizeof(float));

	int ffts[2];
	int idist;int odist;
	int inembed[2];int onembed[2];
	//fft 2d 
	ffts[0] = 2*N; ffts[1] = 2*N;
	idist = (2*N+2*M)*(2*N+2*M);odist = (2*N+2*M)*(2*N+2*M);
	inembed[0] = 2*N+2*M; inembed[1] = 2*N+2*M;
	onembed[0] = 2*N+2*M; onembed[1] = 2*N+2*M;
	hipfftPlanMany(&plan2d, 2, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_C2C, Nz); 

	//fft 1d	
	ffts[0] = N;
	idist = N;odist = N;
	inembed[0] = N;onembed[0] = N;
	hipfftPlanMany(&plan1d, 1, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_C2C, Ntheta*Nz);
}

radonusfft::~radonusfft()
{	
	hipFree(f);
	hipFree(g);	
	hipFree(fde);
	hipFree(x);
	hipFree(y);
	hipFree(theta);
	hipfftDestroy(plan2d);
	hipfftDestroy(plan1d);
}

void radonusfft::fwdR(float2* g_, float2* f_, float* theta_, hipStream_t s)
{	
	dim3 BS2d(32,32);
	dim3 BS3d(32,32,1);

	dim3 GS2d0(ceil(N/(float)BS2d.x),ceil(Ntheta/(float)BS2d.y));
	dim3 GS3d0(ceil(N/(float)BS3d.x),ceil(N/(float)BS3d.y),ceil(Nz/(float)BS3d.z));
	dim3 GS3d1(ceil(2*N/(float)BS3d.x),ceil(2*N/(float)BS3d.y),ceil(Nz/(float)BS3d.z));
	dim3 GS3d2(ceil((2*N+2*M)/(float)BS3d.x),ceil((2*N+2*M)/(float)BS3d.y),ceil(Nz/(float)BS3d.z));
	dim3 GS3d3(ceil(N/(float)BS3d.x),ceil(Ntheta/(float)BS3d.y),ceil(Nz/(float)BS3d.z));
	hipMemcpyAsync(f,f_,N*N*Nz*sizeof(float2),hipMemcpyDefault,s);
	hipMemcpyAsync(theta,theta_,Ntheta*sizeof(float),hipMemcpyDefault,s);  	
	hipMemsetAsync(fde,0,(2*N+2*M)*(2*N+2*M)*Nz*sizeof(float2),s);
	takexy<<<GS2d0, BS2d,0,s>>>(x,y,theta,N,Ntheta);
	divphi<<<GS3d0, BS3d,0,s>>>(fde,f,mu,M,N,Nz);

	fftshiftc<<<GS3d2, BS3d,0,s>>>(fde,2*N+2*M,Nz);
	hipfftSetStream(plan2d,s);
	hipfftExecC2C(plan2d, (hipfftComplex*)&fde[M+M*(2*N+2*M)],(hipfftComplex*)&fde[M+M*(2*N+2*M)],HIPFFT_FORWARD);
	fftshiftc<<<GS3d2, BS3d,0,s>>>(fde,2*N+2*M,Nz);

	wrap<<<GS3d2, BS3d,0,s>>>(fde,N,Nz,M);
	gather<<<GS3d3, BS3d,0,s>>>(g,fde,x,y,M,mu,N,Ntheta,Nz);
	mulr<<<GS3d3,BS3d,0,s>>>(g,1.0f/(4*N*N*N*sqrt(N*Ntheta)),-(center-N/2),N,Ntheta,Nz);
		
	fftshift1c<<<GS3d3, BS3d,0,s>>>(g,N,Ntheta,Nz);
	hipfftSetStream(plan1d,s);
	hipfftExecC2C(plan1d, (hipfftComplex*)g,(hipfftComplex*)g,HIPFFT_BACKWARD);
	fftshift1c<<<GS3d3, BS3d,0,s>>>(g,N,Ntheta,Nz);

	
	hipMemcpyAsync(g_,g,N*Ntheta*Nz*sizeof(float2),hipMemcpyDefault,s);  	
}

void radonusfft::adjR(float2* f_, float2* g_, float* theta_, bool filter, hipStream_t s)
{
	dim3 BS2d(32,32);
	dim3 BS3d(32,32,1);

	dim3 GS2d0(ceil(N/(float)BS2d.x),ceil(Ntheta/(float)BS2d.y));
	dim3 GS3d0(ceil(N/(float)BS3d.x),ceil(N/(float)BS3d.y),ceil(Nz/(float)BS3d.z));
	dim3 GS3d1(ceil(2*N/(float)BS3d.x),ceil(2*N/(float)BS3d.y),ceil(Nz/(float)BS3d.z));
	dim3 GS3d2(ceil((2*N+2*M)/(float)BS3d.x),ceil((2*N+2*M)/(float)BS3d.y),ceil(Nz/(float)BS3d.z));
	dim3 GS3d3(ceil(N/(float)BS3d.x),ceil(Ntheta/(float)BS3d.y),ceil(Nz/(float)BS3d.z));

	hipMemcpyAsync(g,g_,N*Ntheta*Nz*sizeof(float2),hipMemcpyDefault,s);
	hipMemcpyAsync(theta,theta_,Ntheta*sizeof(float),hipMemcpyDefault,s);  	

	hipMemsetAsync(fde,0,(2*N+2*M)*(2*N+2*M)*Nz*sizeof(float2),s);

	takexy<<<GS2d0, BS2d,0,s>>>(x,y,theta,N,Ntheta);

	fftshift1c<<<GS3d3, BS3d,0,s>>>(g,N,Ntheta,Nz);
	hipfftSetStream(plan1d,s);
	hipfftExecC2C(plan1d, (hipfftComplex*)g,(hipfftComplex*)g,HIPFFT_FORWARD);
	fftshift1c<<<GS3d3, BS3d,0,s>>>(g,N,Ntheta,Nz);

	if(filter) applyfilter<<<GS3d3, BS3d,0,s>>>(g,N,Ntheta,Nz);	
	mulr<<<GS3d3,BS3d,0,s>>>(g,1.0f/(4*N*N*N*sqrt(N*Ntheta)),(center-N/2),N,Ntheta,Nz);

	scatter<<<GS3d3, BS3d,0,s>>>(fde,g,x,y,M,mu,N,Ntheta,Nz);
	wrapadj<<<GS3d2, BS3d,0,s>>>(fde,N,Nz,M);

	fftshiftc<<<GS3d2, BS3d,0,s>>>(fde,2*N+2*M,Nz);
	hipfftSetStream(plan2d,s);
	hipfftExecC2C(plan2d, (hipfftComplex*)&fde[M+M*(2*N+2*M)],(hipfftComplex*)&fde[M+M*(2*N+2*M)],HIPFFT_BACKWARD);
	fftshiftc<<<GS3d2, BS3d,0,s>>>(fde,2*N+2*M,Nz);

	unpaddivphi<<<GS3d0, BS3d,0,s>>>(f,fde,mu,M,N,Nz);
	// mulr<<<GS3d0,BS3d,0,s>>>(f,1.0f/(4*N*N*N*sqrt(N*Ntheta)),N,N,Nz);

	hipMemcpyAsync(f_,f,N*N*Nz*sizeof(float2),hipMemcpyDefault,s);
}